#include "hip/hip_runtime.h"
#include <stdio.h>

#include "reduction.cuh"

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

// cuda thread synchronization
__global__ void grid_stride_reduction_kernel(float* d_out, float* d_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    // Calculate input with grid-stride loop and save to shared memory
    float input = 0.f;
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x)
    {
        input += d_in[i];
    }
    s_data[threadIdx.x] = input;

    __syncthreads();

    // Perform reduction with sequential addressing
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }

        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        d_out[blockIdx.x] = s_data[0];
    }
}

void grid_stride_reduction(float* d_out, float* d_in, int n_threads, unsigned int size)
{
    int num_sms;
    int num_blocks_per_sm;

    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, grid_stride_reduction_kernel, n_threads, n_threads * sizeof(float));
    
    hipMemcpy(d_out, d_in, size * sizeof(float), hipMemcpyDeviceToDevice);

    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1) / n_threads);

    grid_stride_reduction_kernel<<<n_blocks,n_threads,n_threads*sizeof(float),0>>>(d_out, d_out, size);
    grid_stride_reduction_kernel<<<1,n_threads,n_threads*sizeof(float),0>>>(d_out, d_out, n_blocks);
}